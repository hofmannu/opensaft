#include "hip/hip_runtime.h"
#include "color_mapper.cuh"

void color_mapper::set_maxVal(const float _maxVal){
	maxVal = _maxVal;
	calc_max_abs();
	return;
}

void color_mapper::set_minVal(const float _minVal){
	minVal = _minVal;
	calc_max_abs();
	return;
}

void color_mapper::set_minCol(const float* _minCol){
	for (uint8_t iCol = 0; iCol < 4; iCol++)
		minCol[iCol] = _minCol[iCol];

	calc_span_col();
	return;
}

void color_mapper::set_maxCol(const float* _maxCol){
	for (uint8_t iCol = 0; iCol < 4; iCol++)
		maxCol[iCol] = _maxCol[iCol];

	calc_span_col();
	return;
}

void color_mapper::calc_span_col()
{
	for (uint8_t idx = 0; idx < 4; idx++)
		spanCol[idx] = maxCol[idx] - minCol[idx];

	return;
}

void color_mapper::calc_max_abs()
{
	maxAbsVal = (abs(maxVal) > abs(minVal)) ? abs(maxVal) : abs(minVal);
	return;
}


void color_mapper::convert_to_rgba(
	const float* dataIn,
	const uint64_t nElem,
	unsigned char* dataOut) const
{
	
	float spanTemp = maxVal - minVal;
	float spanColTemp[4];
	for (unsigned int iCol = 0; iCol < 4; iCol++)
		spanColTemp[iCol] = maxCol[iCol] - minCol[iCol];

	// scale whole array to range from to 0 to 1
	float temp;
	for (uint64_t iElem = 0; iElem < nElem; iElem++){
		temp = (dataIn[iElem] - minVal) / spanTemp; // scale to [0 ... 1]
		
		// limit temp to 0 ... 1
		if (temp < 0)
			temp = 0;
		else if (temp > 1)
			temp = 1;

		#pragma unroll
		for (unsigned char iCol = 0; iCol < 4; iCol++)
			dataOut[iCol + iElem * 4] = 
				(minCol[iCol] + temp * spanColTemp[iCol]) * 255;
	}

	return;
}

// make diverging colormap centered around 0 or 1
void color_mapper::convert_to_divmap(
	const float* dataIn,
	const uint64_t nElem,
	unsigned char* dataOut
	) const
{
	// scale whole array to range from to 0 to 1
	float temp;
	for (uint64_t iElem = 0; iElem < nElem; iElem++){
		temp = abs(dataIn[iElem]) / maxAbsVal; // scale to [0 ... 1]
		if (temp > 1)
			temp = 1;

		if (dataIn[iElem] < 0)
		{
			#pragma unroll
			for (unsigned char iCol = 0; iCol < 4; iCol++)
			{
				dataOut[iCol + iElem * 4] = (1 - temp * minCol[iCol]) * 255;
			}

		}
		else
		{
			#pragma unroll
			for (unsigned char iCol = 0; iCol < 4; iCol++)
				dataOut[iCol + iElem * 4] = (1 - temp * maxCol[iCol]) * 255;
		}
		dataOut[4] = 255;
		
	}
	return;
}

void color_mapper::convert_to_map(
	const float* dataIn,
	const uint64_t nElem,
	unsigned char* dataOut) const
{
	if (mapType == 0)
		convert_to_rgba(dataIn, nElem, dataOut);
	else if (mapType == 1)
	{
		convert_to_divmap(dataIn, nElem, dataOut);
		// set default colors for diverging map

	}
	return;
}

void color_mapper::set_mapType(const uint8_t _mapType)
{
	mapType = _mapType;
	if (mapType == 1)
	{
		// set minVal to bright blue
		minCol[0] = 1.0; // r
		minCol[1] = 1.0; // g
		minCol[2] = 0.0; // b
		minCol[3] = 0.0; // alpha
		// set maxVal to bright red
		maxCol[0] = 0.0;
		maxCol[1] = 1.0;
		maxCol[2] = 1.0;
		maxCol[3] = 0.0;	
	}
	return;
}