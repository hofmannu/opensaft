#include "hip/hip_runtime.h"
#include "saft.cuh"

/*
	main saft kernel

	todo:
	- implement boundary calculation
	- implement frequency band filtering
*/

__global__ void SAFT
(
	float* outputVol, // reconstructed output volume [it, ix, iy]
	const float* inputVol, // input signal matrix [it, ix, iy]
	const int nT, // number of elemetns in time
	const int nX, // number of elements along x
	const int nY, // number of elements along y
	const float dt, // resolution in time domain
	const float dx, // resolution along x [m]
	const float dy, // resolution along y [m]
	const float t0, // time of first index
	const float c0, // speed of sound in medium [m/s]
	const float critRatio, // critical ratio betwen rDist and deltaZ
	const float fd, // focal distance
	const float rMin // this is the minimum size accepted for any angle
	)
{
	// get index in output volume
	const int zIm = blockIdx.x * blockDim.x + threadIdx.x;
	const int xIm = blockIdx.y * blockDim.y + threadIdx.y;
	const int yIm = blockIdx.z * blockDim.z + threadIdx.z;

	if ((xIm < nX) && (yIm < nY) && (zIm < nT))
	{
		const float zDepth = (t0 + dt * (float) zIm) * c0; // depth of reconstructed point 
		
		// 1 if far field, -1 if close field
		const float signMultip = -1 + 2 * ((float) (zDepth >= fd));
		
		const float deltaZ = zDepth - fd; // z distance between focal point and reson point
		float rfsaft = 0; // coherent saft sum
		float rfsaftabs = 0; // incoherent saft sum
		const int idxFoc = (fd / c0 - t0) / dt + 0.5; // t idx of focal point

		// limit reconstruction range to a local subfield
		const int yConsider = critRatio * fabsf(deltaZ) / dy; // index range to consider in y
		const int xConsider = critRatio * fabsf(deltaZ) / dx; // index range to consider in x
		const int yStartIdx = ((yIm - yConsider) <= 0) ? 0 : yIm - yConsider;
		const int xStartIdx = ((xIm - xConsider) <= 0) ? 0 : xIm - xConsider;
		const int yStopIdx = ((yIm + yConsider) >= nY) ? nY - 1 : yIm + yConsider;
		const int xStopIdx = ((xIm + xConsider) >= nX) ? nX - 1 : xIm + xConsider;

		int nElem = 0;

		#pragma unroll
		for (int iY = yStartIdx; iY <= yStopIdx; iY++) // run over all other a scans in y
		{
			const float yRel = dy * (float) (iY - yIm);  
			#pragma unroll
			for (int iX = xStartIdx; iX <= xStopIdx; iX++) // run over all other a scans in x
			{
				const float xRel = dx * (float) (iX - xIm);
				const float rDist = sqrtf(xRel * xRel + yRel * yRel);
				const float currRatio = rDist / fabsf(deltaZ);

				if (currRatio <= critRatio)
				{
					// calculate distance, delay time, and index
					const float distTot = sqrtf(rDist * rDist + deltaZ * deltaZ);
					const int deltaT = distTot / c0 / dt + 0.5;
					const int tIdx = idxFoc + deltaT * signMultip; 
					
					if ((tIdx >= 0) && (tIdx < nT))
					{
						const int dataIdx = tIdx + nT * (iX + nX * iY);
						rfsaft += inputVol[dataIdx];
						rfsaftabs += fabsf(inputVol[dataIdx]);
						nElem++;
					} 
				}
			}
		}

		if (rfsaftabs > 0)
		{
			const float cf = rfsaft * rfsaft / 
				(rfsaftabs * rfsaftabs * ((float) nElem));
			outputVol[zIm + nT * (xIm + nX * yIm)] = signMultip * rfsaft * cf;
			
			// if (zDepth > fd)
			// 	outputVol[zIm + nT * (xIm + nX * yIm)] = rfsaft * cf;
			// else
			// 	outputVol[zIm + nT * (xIm + nX * yIm)] = -rfsaft * cf;
		}
		else
		{
			outputVol[zIm + nT * (xIm + nX * yIm)] = 0;
		}


	}

	return;
}

void saft::crop()
{
	// check if all max values are bigger then the minimum values, otherwise swap them
	sett.sortCropping();

	// find index closest to minimum cropping positions
	uint64_t idxMin[3];
	uint64_t idxMax[3];
	for (uint8_t iDim = 0; iDim < 3; iDim++)
	{
		idxMin[iDim] = preprocData.getIdx(sett.get_cropMin(iDim) * 1e-3, iDim);
		// printf("min idx along %d is %d \n", iDim, idxMin[iDim]);
		// find index closest to maximum cropping positions
		idxMax[iDim] = preprocData.getIdx(sett.get_cropMax(iDim) * 1e-3, iDim);
		// printf("max idx alomg %d is %d \n", iDim, idxMax[iDim]);
	}

	// push res origin and size over from original dataset
	for (uint8_t iDim = 0; iDim < 3; iDim++)
	{
		
		croppedData.set_origin(iDim, // push origin 
			preprocData.get_pos(idxMin[iDim], iDim));
		
		croppedData.set_res(iDim, // push resolution
			preprocData.get_res(iDim));
		// push new size
		croppedData.set_dim(iDim, idxMax[iDim] - idxMin[iDim] + 1);
	}
	croppedData.alloc_memory();

	// fill freshly allocated memory
	uint64_t runIdx = 0;
	#pragma unroll
	for (uint64_t iy = idxMin[2]; iy < idxMax[2]; iy++)
	{
		const uint64_t iyRel = iy - idxMin[2];
		#pragma unroll
		for (uint64_t ix = idxMin[1]; ix < idxMax[1]; ix++)
		{
			const uint64_t ixRel = ix - idxMin[1];
			#pragma unroll
			for (uint64_t it = idxMin[0]; it < idxMax[0]; it++)
			{
				const uint64_t itRel = it - idxMin[0];
				croppedData.set_value(itRel, ixRel, iyRel, preprocData.get_value(it, ix, iy));
				runIdx++;
			}
		}
	}

	return;
}

void saft::recon()
{
	// perform cropping of dataset to user defined boundaries
	crop();
	
	// for improved readability
	const int nt = croppedData.get_dim(0);
	const int nx = croppedData.get_dim(1); 
	const int ny = croppedData.get_dim(2);

	float* outputVol_dev;
	float* inputVol_dev;

	// allocate memory on GPU
	bool m1 = (hipSuccess != hipMalloc( (void**)&inputVol_dev, 
		croppedData.get_nElements() * sizeof(float) ));
	m1 |= (hipSuccess != hipMalloc( (void**)&outputVol_dev, 
		croppedData.get_nElements() * sizeof(float) ));
	if (m1)
	{
		printf("Could not allocate memory on GPU");
		throw "CudaMemAllocErr";
		return;
	}

	// copy preprocessed volume to gpu
	// copy signal matrix over to GPU and check if successful
	bool cpy1 = (hipSuccess != hipMemcpy(inputVol_dev, croppedData.get_pdata(), 
			croppedData.get_nElements() * sizeof(float), hipMemcpyHostToDevice));
	if (cpy1)
	{
		printf("Could not copy array to GPU");
		throw "CudaMemCpyErr";	
		return;
	}

	// define kernel size
	dim3 blockSize(1, 16, 16);
	dim3 gridSize(
				(nt + blockSize.x - 1) / blockSize.x, 
				(nx + blockSize.y - 1) / blockSize.y, 
				(ny + blockSize.z - 1) / blockSize.z);

	const float deltaZT = sqrt(
		trans.get_focalDistance() * trans.get_focalDistance() -
		trans.get_rAperture() * trans.get_rAperture());
	const float critRatio = trans.get_rAperture() / deltaZT; 

	// execute actual kernel
	// all in SI units!
	SAFT<<<  gridSize, blockSize >>>
	(
		outputVol_dev,
		(const float*) inputVol_dev,
		nt, nx, ny,
		croppedData.get_res(0), croppedData.get_res(1), croppedData.get_res(2),
		croppedData.get_origin(0), 
		sett.get_sos(), // speed of sound in medium
		critRatio,
		trans.get_focalDistance() * 1e-3, // fical distance of transducer [m]
		sett.get_rMin()
		);
	hipDeviceSynchronize();

	// check if kernel execution was successful
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("error during kernel execution:\n");
		printf(hipGetErrorString(err));
		throw "CudaKernelCrash";
	}

	for (uint8_t iDim = 0; iDim < 3; iDim++)
	{
		reconData.set_dim(iDim, croppedData.get_dim(iDim));
		reconData.set_res(iDim, croppedData.get_res(iDim));
		reconData.set_origin(iDim, croppedData.get_origin(iDim));
	}
	reconData.set_res(0, croppedData.get_res(0) * sett.get_sos());
	reconData.set_origin(0, croppedData.get_origin(0) * sett.get_sos());
	
	reconData.alloc_memory();

	hipMemcpy(reconData.get_pdata(), outputVol_dev, 
		reconData.get_nElements() * sizeof(float), hipMemcpyDeviceToHost );

	// free gpu memory again
	hipFree(outputVol_dev);
	hipFree(inputVol_dev);

	// calculate minimum and maximum value in reconstructed volume
	reconData.calcMinMax();
	return;

}