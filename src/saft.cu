#include "hip/hip_runtime.h"
#include "saft.cuh"

// gpu kernel for SAFT
__global__ void
SAFT(float* outputVol,        // reconstructed output volume [it, ix, iy]
     const float* inputVol,   // input signal matrix [it, ix, iy]
     const int nT,            // number of elemetns in time
     const int nX,            // number of elements along x
     const int nY,            // number of elements along y
     const float dt,          // resolution in time domain
     const float dx,          // resolution along x [m]
     const float dy,          // resolution along y [m]
     const float t0,          // time of first index
     const float c0,          // speed of sound in medium [m/s]
     const float critRatio,   // critical ratio betwen rDist and deltaZ
     const float fd,          // focal distance
     const float rMin,        // this is the minimum size accepted for any angle
     const bool flagCfWeight, // should we do coherence factor weighting?
     volatile int* progressCounter) {
  // get index in output volume
  const int zIm = blockIdx.x * blockDim.x + threadIdx.x;
  const int xIm = blockIdx.y * blockDim.y + threadIdx.y;
  const int yIm = blockIdx.z * blockDim.z + threadIdx.z;

  if ((xIm < nX) && (yIm < nY) && (zIm < nT)) {
    const float zDepth =
        (t0 + dt * (float)zIm) * c0; // depth of reconstructed point

    // 1 if far field, -1 if close field
    const int signMultip = -1 + 2 * ((float)(zDepth >= fd));

    const float deltaZ =
        zDepth - fd;     // z distance between focal point and reson point
    float rfsaft = 0;    // coherent saft sum
    float rfsaftabs = 0; // incoherent saft sum
    const int idxFoc = (fd / c0 - t0) / dt + 0.5; // t idx of focal point

    // limit reconstruction range to a local subfield
    const int yConsider =
        critRatio * fabsf(deltaZ) / dy; // index range to consider in y
    const int xConsider =
        critRatio * fabsf(deltaZ) / dx; // index range to consider in x
    const int yStartIdx = ((yIm - yConsider) <= 0) ? 0 : yIm - yConsider;
    const int xStartIdx = ((xIm - xConsider) <= 0) ? 0 : xIm - xConsider;
    const int yStopIdx = ((yIm + yConsider) >= nY) ? nY - 1 : yIm + yConsider;
    const int xStopIdx = ((xIm + xConsider) >= nX) ? nX - 1 : xIm + xConsider;

    int nElem = 0;

#pragma unroll
    for (int iY = yStartIdx; iY <= yStopIdx;
         iY++) // run over all other a scans in y
    {
      const float yRel = dy * ((float)(iY - yIm));
#pragma unroll
      for (int iX = xStartIdx; iX <= xStopIdx;
           iX++) // run over all other a scans in x
      {
        const float xRel = dx * ((float)(iX - xIm));
        const float rDist = sqrtf(xRel * xRel + yRel * yRel);
        const float currRatio = rDist / fabsf(deltaZ);

        if ((currRatio <= critRatio) || (rDist < rMin)) {
          // calculate distance, delay time, and index
          const float distTot = sqrtf(rDist * rDist + deltaZ * deltaZ);
          const int deltaT = distTot / c0 / dt + 0.5;
          const int tIdx = idxFoc + deltaT * signMultip;

          if ((tIdx >= 0) && (tIdx < nT)) {
            const int dataIdx = tIdx + nT * (iX + nX * iY);
            rfsaft += inputVol[dataIdx];
            rfsaftabs += fabsf(inputVol[dataIdx]);
            nElem++;
          }
        }
      }
    }

    if (rfsaftabs > 0) {
      const float cf =
          flagCfWeight
              ? (rfsaft * rfsaft / (rfsaftabs * rfsaftabs * ((float)nElem)))
              : 1.0;
      outputVol[zIm + nT * (xIm + nX * yIm)] =
          ((float)signMultip) * rfsaft * cf;

    } else {
      outputVol[zIm + nT * (xIm + nX * yIm)] = 0;
    }
    // increase counter of reconstructed points
    atomicAdd((int*)progressCounter, 1);
    __threadfence_system();
  }

  return;
}

// run the kernel but on the CPU
void saft::saft_cpu() {

  // declare a few varaibles for improved readabilit
  const uint32_t nT = croppedData.get_dim(0);
  const uint32_t nX = croppedData.get_dim(1);
  const uint32_t nY = croppedData.get_dim(2);

  // for improved readability
  const float t0 = croppedData.get_origin(0);
  const float dt = croppedData.get_res(0);
  const float dx = croppedData.get_res(1);
  const float dy = croppedData.get_res(2);
  const float c0 = sett.get_flagUs() ? 0.5 * sett.get_sos() : sett.get_sos();
  const float fd = trans.get_focalDistance() * 1e-3;

  float* outputVol = reconData.get_pdata();
  const float* inputVol = croppedData.get_pdata();

  const float deltaZT =
      sqrt(trans.get_focalDistance() * trans.get_focalDistance() -
           trans.get_rAperture() * trans.get_rAperture());
  const float critRatio = trans.get_rAperture() / deltaZT;

  // outer loop defining that we do everything for each and every pixel
  for (uint32_t yIm = 0; yIm < nY; yIm++) {
    for (uint32_t xIm = 0; xIm < nX; xIm++) {
      for (uint32_t zIm = 0; zIm < nT; zIm++) {
        const float zDepth =
            (t0 + dt * (float)zIm) * c0; // depth of reconstructed point

        // 1 if far field, -1 if close field
        const float signMultip = -1 + 2 * ((float)(zDepth >= fd));

        const float deltaZ =
            zDepth - fd;     // z distance between focal point and reson point
        float rfsaft = 0;    // coherent saft sum
        float rfsaftabs = 0; // incoherent saft sum
        const int idxFoc = (fd / c0 - t0) / dt + 0.5; // t idx of focal point

        // limit reconstruction range to a local subfield
        const int yConsider =
            critRatio * fabsf(deltaZ) / dy; // index range to consider in y
        const int xConsider =
            critRatio * fabsf(deltaZ) / dx; // index range to consider in x
        const int yStartIdx = ((yIm - yConsider) <= 0) ? 0 : yIm - yConsider;
        const int xStartIdx = ((xIm - xConsider) <= 0) ? 0 : xIm - xConsider;
        const int yStopIdx =
            ((yIm + yConsider) >= nY) ? nY - 1 : yIm + yConsider;
        const int xStopIdx =
            ((xIm + xConsider) >= nX) ? nX - 1 : xIm + xConsider;

        int nElem = 0;

#pragma unroll
        for (int iY = yStartIdx; iY <= yStopIdx;
             iY++) // run over all other a scans in y
        {
          const float yRel = dy * (float)(iY - yIm);
#pragma unroll
          for (int iX = xStartIdx; iX <= xStopIdx;
               iX++) // run over all other a scans in x
          {
            const float xRel = dx * (float)(iX - xIm);
            const float rDist = sqrtf(xRel * xRel + yRel * yRel);
            const float currRatio = rDist / fabsf(deltaZ);

            if (currRatio <= critRatio) {
              // calculate distance, delay time, and index
              const float distTot = sqrtf(rDist * rDist + deltaZ * deltaZ);
              const int deltaT = distTot / c0 / dt + 0.5;
              const int tIdx = idxFoc + deltaT * signMultip;

              if ((tIdx >= 0) && (tIdx < nT)) {
                const int dataIdx = tIdx + nT * (iX + nX * iY);
                rfsaft += inputVol[dataIdx];
                rfsaftabs += fabsf(inputVol[dataIdx]);
                nElem++;
              }
            }
          }
        }

        if (rfsaftabs > 0) {
          const float cf =
              sett.get_flagCoherenceW()
                  ? rfsaft * rfsaft / (rfsaftabs * rfsaftabs * ((float)nElem))
                  : 1;
          outputVol[zIm + nT * (xIm + nX * yIm)] = signMultip * rfsaft * cf;

          // if (zDepth > fd)
          // 	outputVol[zIm + nT * (xIm + nX * yIm)] = rfsaft * cf;
          // else
          // 	outputVol[zIm + nT * (xIm + nX * yIm)] = -rfsaft * cf;
        } else {
          outputVol[zIm + nT * (xIm + nX * yIm)] = 0;
        }

        // update percDone
        const uint32_t nTotal = nT * nY * nX;
        const uint32_t nCurr = zIm + nT * (xIm + yIm * nX);
        percDone = ((float)nCurr) / ((float)nTotal) * 100;
      }
    }
  }

  return;
}

// start the actual reconstruction
void saft::recon() {
  isRunning = 1;
  crop();      // perform cropping of dataset to user defined boundaries
  remove_dc(); // remove dc component of each individual a scan in croppedData

  // push properties over to reconstructed dataset and allocate memory
  for (uint8_t iDim = 0; iDim < 3; iDim++) {
    reconData.set_dim(iDim, croppedData.get_dim(iDim));
    reconData.set_res(iDim, croppedData.get_res(iDim));
    reconData.set_origin(iDim, croppedData.get_origin(iDim));
  }

  // overwrite resolution and origin in z for us pulse echo mode
  const float dz = sett.get_flagUs()
                       ? croppedData.get_res(0) * sett.get_sos() / 2
                       : croppedData.get_res(0) * sett.get_sos();
  reconData.set_res(0, dz);
  const float originZ = sett.get_flagUs()
                            ? croppedData.get_origin(0) * sett.get_sos() / 2
                            : croppedData.get_origin(0) * sett.get_sos();
  reconData.set_origin(0, originZ);

  reconData.alloc_memory();                  // allocate memory
  tStart = std::chrono::system_clock::now(); // save start time to variable

  if (sett.get_flagGpu()) {
    // for improved readability
    const int nt = croppedData.get_dim(0);
    const int nx = croppedData.get_dim(1);
    const int ny = croppedData.get_dim(2);

    float* outputVol_dev;
    float* inputVol_dev;

    // allocate memory on GPU
    bool m1 = (hipSuccess !=
               hipMalloc((void**)&inputVol_dev,
                          croppedData.get_nElements() * sizeof(float)));
    m1 |= (hipSuccess !=
           hipMalloc((void**)&outputVol_dev,
                      croppedData.get_nElements() * sizeof(float)));
    if (m1) {
      printf("Could not allocate memory on GPU\n");
      throw "CudaMemAllocErr";
      return;
    }

    // copy preprocessed volume to gpu
    // copy signal matrix over to GPU and check if successful
    const bool cpy1 =
        (hipSuccess != hipMemcpy(inputVol_dev, croppedData.get_pdata(),
                                   croppedData.get_nElements() * sizeof(float),
                                   hipMemcpyHostToDevice));
    if (cpy1) {
      printf("Could not copy array to GPU\n");
      throw "CudaMemCpyErr";
      return;
    }

    // define kernel size
    const dim3 blockSize(1, 16, 16);
    dim3 gridSize((nt + blockSize.x - 1) / blockSize.x,
                  (nx + blockSize.y - 1) / blockSize.y,
                  (ny + blockSize.z - 1) / blockSize.z);

    const float deltaZT =
        sqrt(trans.get_focalDistance() * trans.get_focalDistance() -
             trans.get_rAperture() * trans.get_rAperture());
    const float critRatio = trans.get_rAperture() / deltaZT;

    // if the dataset is ultrasound pulse echo measurement, just multiply sos
    // with 0.5
    const float reconSos =
        sett.get_flagUs() ? (sett.get_sos() * 0.5) : sett.get_sos();

    volatile int *d_progress, *h_progress;
    const bool map1 = (hipSuccess != hipSetDeviceFlags(hipDeviceMapHost));
    if (map1) {
      printf("Something went wrong while mapping");
      throw "CudaError";
      return;
    }

    const bool m2 =
        (hipSuccess !=
         hipHostAlloc((void**)&h_progress, sizeof(int), hipHostMallocMapped));
    if (m2) {
      printf("Something went wrong while allocating the progress integer");
      throw "CudaMemAllocErr";
      return;
    }

    const bool mPtr =
        (hipSuccess !=
         hipHostGetDevicePointer((int**)&d_progress, (int*)h_progress, 0));
    if (mPtr) {
      printf("Something went wrong while getting the device pointer");
      throw "CudaError";
      return;
    }

    *h_progress = 0;

    // execute actual kernel
    // all in SI units!
    SAFT<<<gridSize, blockSize>>>(
        outputVol_dev, (const float*)inputVol_dev, nt, nx, ny,
        croppedData.get_res(0), croppedData.get_res(1), croppedData.get_res(2),
        croppedData.get_origin(0),
        reconSos, // speed of sound in medium
        critRatio,
        trans.get_focalDistance() * 1e-3, // fical distance of transducer [m]
        sett.get_rMin(), sett.get_flagCoherenceW(), d_progress);

    int value = 0;
    do {
      int value1 = *h_progress;
      if (value1 > value) {
        // printf("h_progress = %d\n", value1);
        value = value1;
        percDone = ((float)value) / ((float)nx * ny * nt) * 100.0;
        auto tCurr = std::chrono::system_clock::now();
        const double tPassed =
            std::chrono::duration_cast<std::chrono::seconds>(tCurr - tStart)
                .count();
        tRemain = tPassed / percDone * (100.0 - percDone);
      }
    } while (value < (nt * nx * ny));

    hipDeviceSynchronize();

    // check if kernel execution was successful
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      printf("error during kernel execution:\n");
      printf(hipGetErrorString(err));
      printf("\n");
      throw "CudaKernelCrash";
    }
    hipMemcpy(reconData.get_pdata(), outputVol_dev,
               reconData.get_nElements() * sizeof(float),
               hipMemcpyDeviceToHost);

    // free gpu memory again
    hipFree(outputVol_dev);
    hipFree(inputVol_dev);
    isRunning = 0;
  } else {
    isRunning = 1;
    saft_cpu();
    isRunning = 0;
  }

  tEnd = std::chrono::system_clock::now();
  reconTime =
      std::chrono::duration_cast<std::chrono::seconds>(tEnd - tStart).count();
  printf("[saft] reconstruction took %.1f seconds\n", reconTime);

  // calculate minimum and maximum value in reconstructed volume
  reconData.calcMinMax();
  reconData.calcMips();
  return;
}

// return a thread which runs the reconstruction
std::thread saft::recon2thread() {
  return std::thread([=] { recon(); });
}